#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
// #include <cutil.h>
#include ""
#include "hip/device_functions.h"
#include ""
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <iostream>
using namespace std;

#define DATATYPE int
#define ARRAYLEN 128 * 1024 * 1024

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        std::cerr << std::endl
                  << " CUDA error   " << file
                  << "(" << line << ")"
                  << " : " << errorMessage
                  << " -> " << hipGetErrorString(err) << "(" << (int)err
                  << ") " << std::endl
                  << std::endl;
        hipDeviceReset();
        std::exit(EXIT_FAILURE);
    }
}

#define __CUDA_ERROR(msg)                            \
    {                                                \
        hipDeviceSynchronize();                     \
        __getLastCudaError(msg, __FILE__, __LINE__); \
    }

#define size 100

__constant__ int *prt[size];

__global__ void matrix_add()
{
    for (int i = threadIdx.x; i < size; i += blockDim.x)
    {
        *(prt[i]) = 2;
    }
}

int main()
{

    //   int device;
    //   hipGetDevice(&device);
    hipSetDevice(5);
    int *p = (int *)malloc(sizeof(int) * size);
    printf("host:%d,%d\n", sizeof(int *), sizeof(int));
    // int *pd[size];
    // for (int i = 0; i < size; i++)
    // {
    //     hipMalloc(&(pd[i]), sizeof(int) * 1);
    // }
    int *pd;
    hipMalloc(&pd, sizeof(int) * size);

    int *pdarray[size];
    for (int i = 0; i < size; i++)
    {
        pdarray[i] = pd + i;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(prt), pdarray, sizeof(int *) * size);
    matrix_add<<<1, 256>>>();
    //  __CUDA_ERROR("dfa");
    hipMemcpy(p, pd, sizeof(int) * size, hipMemcpyDeviceToHost);
    // hipMemcpyFromSymbol(hd, HIP_SYMBOL(data), sizeof(int) * 1024);
    // for (int i = 0; i < size; i++)
    // {
    //     hipMemcpy(&p[i], pd[i], sizeof(int) * 1, hipMemcpyDeviceToHost);
    // }
    for (int i = 0; i < size; i++)
    {
        cout << p[i] << " ";
    }
}