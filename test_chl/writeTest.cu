#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
// #include <cutil.h>
#include ""
#include "hip/device_functions.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>
#include <time.h>

#include <stdio.h>
#include <iostream>

using namespace cooperative_groups;


typedef int mytype;
#define ARRAYLEN 32 * 10000000

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		std::cerr << std::endl << " CUDA error   " << file
                  << "(" << line << ")" << " : " << errorMessage
                  << " -> " << hipGetErrorString(err) << "(" << (int) err
                  << ") "<< std::endl << std::endl;
		hipDeviceReset();
		std::exit(EXIT_FAILURE);
	}
}

#define __CUDA_ERROR(msg)                                                       \
                    {                                                           \
                        hipDeviceSynchronize();                                \
                        __getLastCudaError (msg, __FILE__, __LINE__);\
                    }

template<int VW_SIZE,typename T>
__device__ __forceinline__
void VWInclusiveScanAdd(thread_block_tile<VW_SIZE>& tile,const T& value,T& sum)
{
    sum = value;
    for(int i=1;i<=tile.size()/2;i*=2)
    {
        T n = tile.shfl_up(sum, i);
        if (tile.thread_rank() >= i)
        {
            sum += n;
        }
    }
}

template<int VW_SIZE,typename T>
__device__ __forceinline__
void VWWrite(thread_block_tile<VW_SIZE>& tile,int *pAllsize,T* writeStartAddr,
        const int& writeCount,T* data)
{
    int sum = 0;
    int bias = 0;
    VWInclusiveScanAdd<VW_SIZE,int>(tile,writeCount,sum);
    if(tile.thread_rank() == tile.size()-1 && sum !=0)
    {
        bias = atomicAdd(pAllsize,sum);
    }
    bias = tile.shfl(bias,tile.size()-1);
    sum -= writeCount;
    for(int it = 0;it<writeCount;it++)
    {
        *(writeStartAddr+bias+sum+it) = data[it];   
    }
}

template<int VW_SIZE,typename T>
__device__ __forceinline__
void VWWrite_v2(thread_block_tile<VW_SIZE>& tile,int *pAllsize,T* writeStartAddr,
        const int& writeCount,T* data)
{
    int sum = 0;
    int bias = 0;
    int all = 0;
    int k=0;
    int count=writeCount;
    VWInclusiveScanAdd<VW_SIZE,int>(tile,writeCount,sum);
    if(tile.thread_rank() == tile.size()-1 && sum !=0)
    {
        bias = atomicAdd(pAllsize,sum);
    }
    bias = tile.shfl(bias,tile.size()-1);
    while(1)
    {
        unsigned mask = tile.ballot(count>0);
        int flag = __ffs(mask);
        if (flag  == 0)
            break;
        if(count > 0)
        {
            unsigned mymask = 1 << (tile.thread_rank()+1) - 1;
            mymask = mask & mymask ;
            int mybias = __ffs(mymask);
            *(writeStartAddr+bias+all+mybias) = data[k++];
            count--;
        }
    }
}

template<int VW_SIZE,typename T>
__device__ __forceinline__
void VWWrite_v3(thread_block_tile<VW_SIZE>& tile,int *pAllsize,T* writeStartAddr,
        const int& writeCount,T* data)
{
    int sum = 0;
    int bias = 0;
    VWInclusiveScanAdd<VW_SIZE,int>(tile,writeCount,sum);
    if(tile.thread_rank() == tile.size()-1 && sum !=0)
    {
        bias = atomicAdd(pAllsize,sum);
    }
    bias = tile.shfl(bias,tile.size()-1);
    sum -= writeCount;
    for(int it = 0;it<8;it++)
    {
        *(writeStartAddr+bias+sum+it) = data[it];     
    }
}

template<int VW_SIZE,typename T>
__device__ __forceinline__
void VWWrite_v4(thread_block_tile<VW_SIZE>& tile,int *pAllsize,T* writeStartAddr,
        const int& writeCount,T* data)
{
    int sum = 0;
    int bias = 0;
    int all = 0;
    int k=0;
    int count=writeCount;
    VWInclusiveScanAdd<VW_SIZE,int>(tile,writeCount,sum);
    if(tile.thread_rank() == tile.size()-1 && sum !=0)
    {
        bias = atomicAdd(pAllsize,sum);
    }
    bias = tile.shfl(bias,tile.size()-1);
    while(1)
    {
        unsigned mask = tile.ballot(count>0);
        int flag = __ffs(mask);
        if (flag  == 0)
            break;
        if(count > 0)
        {
            unsigned mymask = 1 << (tile.thread_rank()+1) - 1;
            mymask = mask & mymask ;
            int mybias = __ffs(mymask);
#pragma unroll
            for(int i=0;i<8;i++)
            {
                if(k == i)
                {
                    *(writeStartAddr+bias+all+mybias) = data[i];
                }
            }
            k++;
            // *(writeStartAddr+bias+all+mybias) = data[k++];
            count--;
        }
    }
}

__host__ __device__ __forceinline__ int is_write(mytype v)
{
    return !(v%4);
}

#define REG 16

__global__ void test(const mytype *f1,int f1Size, mytype* f2, int* pf2Size)
{
    mytype array[8]={0,1,2,3,4,5,6,7};
    thread_block g = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(g);
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for(;id<f1Size;id=id+stride)
    {
        array[0]=1;
    }
    int k = 8;
    VWWrite<32,mytype>(tile, pf2Size, f2, k , array);
}

__global__ void write_test1(const mytype *f1,int f1Size, mytype* f2, int* pf2Size)
{
    __shared__ mytype array[REG * 512];
    int k=0;
    thread_block g = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(g);
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for(;id<f1Size;id=id+stride)
    {
        mytype v = f1[id];
        if(is_write(v))
        {
            array[threadIdx.x * REG + k++] = v;
        }
        if(tile.any(k>=REG))
        {           
            VWWrite<32,mytype>(tile, pf2Size, f2, k , array + threadIdx.x * REG);
            k=0;
        }       
    }
    VWWrite<32,mytype>(tile, pf2Size, f2, k , array + threadIdx.x * REG);
    k=0;
}

__global__ void write_test2(const mytype *f1,int f1Size, mytype* f2, int* pf2Size)
{
    mytype array[REG];
    int k=0;
    thread_block g = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(g);
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for(;id<f1Size;id=id+stride)
    {
        mytype v = f1[id];
        if(is_write(v))
        {
            array[k++] = v;
        }
        if(tile.any(k>=REG))
        {           
            VWWrite<32,mytype>(tile, pf2Size, f2, k , array);
            k=0;
        }       
    }
    VWWrite<32,mytype>(tile, pf2Size, f2, k , array);
    k=0;
}

__global__ void write_test3(const mytype *f1,int f1Size, mytype* f2, int* pf2Size)
{
    int k=0;
    mytype array[8];
    thread_block g = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(g);
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for(;id<f1Size;id=id+stride)
    {
        mytype v = f1[id];
        int ok = is_write(v);
        if(ok)
        {
            // array[k] = v;
#pragma unroll
            for(int i=0;i<8;i++)
            {
                if(k == i)
                {
                    array[i] = v;
                }
            }
            k++;
        }
        if(tile.any(k>=8))
        {
            VWWrite_v3<32,mytype>(tile, pf2Size, f2, k , array);
            k=0;
        }   
    }
    VWWrite_v3<32,mytype>(tile, pf2Size, f2, k , array);
}

__global__ void write_test4(const mytype *f1,int f1Size, mytype* f2, int* pf2Size)
{
    mytype array[REG];
    int k=0;
    thread_block g = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(g);
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for(;id<f1Size;id=id+stride)
    {
        mytype v = f1[id];
        if(is_write(v))
        {
            array[k++] = v;
        }
        if(tile.any(k>=REG))
        {           
            VWWrite_v2<32,mytype>(tile, pf2Size, f2, k , array);
            k=0;
        }       
    }
    VWWrite_v2<32,mytype>(tile, pf2Size, f2, k , array);
    k=0;
}

#define REG5 1024

__global__ void write_test5(const mytype *f1,int f1Size, mytype* f2, int* pf2Size)
{
    __shared__ int count;
    __shared__ mytype array[REG5];
    __shared__ int all;
    if(threadIdx.x == 0)
        count =0;
    __syncthreads();
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for(;id<f1Size;id=id+stride)
    {
        mytype v = f1[id];
        if(is_write(v))
        {
            int bias = atomicAdd(&count,1);
            array[bias] = v;
        }
        __syncthreads();
        if(count >= REG5 - 512)
        {
            if(threadIdx.x == 0)
                all = atomicAdd(pf2Size,count);
            __syncthreads();
            if(threadIdx.x < count)
                *(f2+all+threadIdx.x) = array[threadIdx.x];
            if(threadIdx.x == 0)
                count =0;
        } 
    }
    if(threadIdx.x == 0)
        all = atomicAdd(pf2Size,count);
    __syncthreads();
    if(threadIdx.x < count)
        *(f2+all+threadIdx.x) = array[threadIdx.x];
}

typedef void(*func)(const mytype*,int, mytype*, int*);

void test_time(func f,const mytype* f1,mytype *f2,int* f2Size)
{
    hipEvent_t s1,s2;
    hipEventCreate(&s1);
    hipEventCreate(&s2);
    hipEventRecord(s1,0);
    hipEventSynchronize(s1);
    f<<<1,512>>>(f1,ARRAYLEN,f2,f2Size);
    hipEventRecord(s2,0);
    hipEventSynchronize(s2);
    float time1;
    hipEventElapsedTime(&time1,s1,s2);
    hipDeviceSynchronize();
    hipEventDestroy(s1);
    hipEventDestroy(s2);
    printf("%f\n",time1);
}
int main()
{
    mytype *f1,*f2;
    int *f2Size;
    srand( (unsigned)time( NULL ) );  
    hipSetDevice(2);
    size_t ds = ARRAYLEN*sizeof(mytype);
    hipMallocManaged(&f1, ds);
    hipMallocManaged(&f2, ds);
    hipMallocManaged(&f2Size, sizeof(int));
    for(int i=0;i<ARRAYLEN;i++)
    {
        f1[i] = rand()%1000;
    }
    int attr = 0;
    hipDeviceGetAttribute(&attr, hipDeviceAttributeConcurrentManagedAccess,0);
    if (attr)
    {
        hipMemPrefetchAsync(f1, ds, 0);
        // hipMemPrefetchAsync(f2, ds, 0);
    }

    int cT=0;
    for(int i=0;i<ARRAYLEN;i++)
    {
        if(is_write(i))
            cT++;
    }

    f2Size[0] = 0;

    test_time(f1,ARRAYLEN,f2,f2Size);

    f2Size[0] = 0;
    
    for(int i=0;i<1;i++)
    {
        f2Size[0] = 0;
        test_time(write_test5,f1,f2,f2Size);
        __CUDA_ERROR("");
    }

    f2Size[0] = 0;
    
    for(int i=0;i<1;i++)
    {
        f2Size[0] = 0;
        test_time(write_test4,f1,f2,f2Size);
    }
    printf("%d,%d\n",f2Size[0],cT);
    
    printf("test2\n");
    for(int i=0;i<1;i++)
    {
        f2Size[0] = 0;
        test_time(write_test3,f1,f2,f2Size);
    }
    printf("%d,%d\n",f2Size[0],cT);

    printf("test3\n");
    for(int i=0;i<1;i++)
    {
        f2Size[0] = 0;
        test_time(write_test2,f1,f2,f2Size);
    }
    printf("%d,%d\n",f2Size[0],cT);

    printf("test4\n");
    for(int i=0;i<1;i++)
    {
        f2Size[0] = 0;
        test_time(write_test1,f1,f2,f2Size);
    }

    __CUDA_ERROR("");

//  for(int i=0;i<f2Size[0];i++)
//     {
//         printf("%d ",f2[i]);
//     }


    printf("%d,%d\n",f2Size[0],cT);
    std::cout << "Success!" << std::endl;
    return 0;
}