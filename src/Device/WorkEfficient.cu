#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------------
Copyright © 2015 by Nicola Bombieri

H-BF is provided under the terms of The MIT License (MIT):

Permission is hereby granted, free of charge, to any person obtaining a copy of
this software and associated documentation files (the "Software"), to deal in
the Software without restriction, including without limitation the rights to
use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
------------------------------------------------------------------------------*/
/**
 * @author Federico Busato
 * Univerity of Verona, Dept. of Computer Science
 * federico.busato@univr.it
 */
#include "Device/HBFGraph.cuh"
#include "XLib.hpp"

#include "Kernels/WorkEfficient_KernelDispath.cu"

namespace
{
template <typename T>
inline bool distanceCompare(dist_t A, T B);

template <>
inline bool distanceCompare<dist_t>(dist_t A, dist_t B)
{
	return A == B;
}

template <>
inline bool distanceCompare<int2>(dist_t A, int2 B)
{
	return A == B.y;
}
} // namespace

void HBFGraph::FrontierDebug(const int FSize, const int level)
{
	if (FSize > max_frontier_size)
		__ERROR("Device memory not sufficient to contain the vertices frontier");
	if (CUDA_DEBUG)
	{
		//__CUDA_ERROR("BellmanFord Host");

		// std::cout << "level: " << level << "\tF2Size: " << FSize << std::endl;
		if (CUDA_DEBUG >= 2)
		{
			if (level <= DEBUG_LEVEL)
			{
				node_t *tmpF1 = new node_t[graph.V * 10];
				hipMemcpy(tmpF1, devF1, FSize * sizeof(node_t), hipMemcpyDeviceToHost);
				printf("\n%s=%d\t", "cuda_frontier_level:", level);
				printExt::host::printArray(tmpF1, FSize, " ");
				delete[] tmpF1;
			}
		}
	}
}

void HBFGraph::WorkEfficient()
{
	int SizeArray[4];
	long long int totalEdges = 0;
	float totalTime = 0;
	std::cout.setf(std::ios::fixed | std::ios::left);

	printf("make degree");

	this->markDegree();

	timer::Timer<timer::HOST> TM_H;
	timer_cuda::Timer<timer_cuda::DEVICE> TM_D;

	unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
	std::default_random_engine generator(seed);
	std::uniform_int_distribution<> distribution(0, graph.V);

	int maxFrontier = std::numeric_limits<int>::min();
	std::vector<int> host_frontiers;
	int *Sources = new int[1];
			if (CHECK_RESULT)
		{
					dynamic_cast<GraphSSSP &>(graph).BellmanFord_Queue_init();
		}
	for (int i = 0; i < N_OF_TESTS; i++)
	{

		Sources[0] = {N_OF_TESTS == 1 ? 0 : distribution(generator)};
		//Sources[0] = 160413;

		int edgeTraversed = graph.E;
		if (CHECK_TRAVERSED_EDGES)
		{
			graph.BFS_Init();
			graph.BFS(Sources[0]);
			edgeTraversed = graph.BFS_visitedEdges();
			graph.BFS_Reset();

			if (edgeTraversed == 0 || (float)graph.E / edgeTraversed < 0.1f)
			{
				i--;
				std::cout << "EdgeTraversed:" << edgeTraversed
						  << " -> Repeat" << std::endl;
				continue;
			}
		}

		//printf("host start\n");

		if (CHECK_RESULT)
		{
			//std::cout << "Computing Host Bellman-Ford..." << std::endl;
			if (CUDA_DEBUG)
			{
				//dynamic_cast<GraphSSSP&>(graph).BoostDijkstra(Sources[0]);
				dynamic_cast<GraphSSSP &>(graph).BellmanFord_Frontier(Sources[0], host_frontiers);
				printf("host_frontiers_size = ");
				for (int i = 0; i < (int)host_frontiers.size(); i++)
					std::cout << " " << host_frontiers[i];
				std::cout << std::endl;
				host_frontiers.resize(0);
			}
			else
			{
				dynamic_cast<GraphSSSP &>(graph).BellmanFord_Queue(Sources[0]);
			}
		}

		//printf("cuda start\n");

		int level = 1, F1Size = 1, F2Size;
		this->init(Sources);

		//======================================================================

		std::vector<int> cuda_frontiers;
		int *last_nodes = new int[this->graph.V];
		for (int i = 0; i < this->graph.V; i++)
			last_nodes[i] = i;
		TM_D.start();

		do
		{
			cuda_frontiers.push_back(F1Size);
			//printf("F1Size:%d",F1Size);
			FrontierDebug(F1Size, level);
			DynamicVirtualWarp(F1Size, level);

			hipMemcpyFromSymbol(SizeArray, HIP_SYMBOL(devF2Size), sizeof(int) * 4);
			F2Size = SizeArray[level & 3];
			F1Size = F2Size;

			level++;
			std::swap<int *>(devF1, devF2);
			maxFrontier = std::max(maxFrontier, F2Size);

		} while (F2Size > 0);
		// if (OUT_DEGREE_OPT)
		// {
		// 	printf("graph.V:%d\n", this->graph.V);

		// 	F1Size = this->graph.V;
		// 	hipMemcpy(devF1, last_nodes, this->graph.V * sizeof(int), hipMemcpyHostToDevice);
		// 	DynamicVirtualWarpForLast(F1Size, level);
		// 	//printf("delete\n");
		// }

		TM_D.stop();
		delete[] last_nodes;
		if (CUDA_DEBUG)
		{
			printf("cuda_frontiers_size = ");
			for (int x : cuda_frontiers)
			{
				printf(" %d", x);
			}
			printf("\n");
		}

		//======================================================================

		float time = TM_D.duration();
		totalTime += time;

		__CUDA_ERROR("BellmanFord Kernel");

		totalEdges += edgeTraversed;
		if (N_OF_TESTS > 1)
			std::cout << "iter: " << std::setw(10) << i
					  << "\ttime: " << std::setw(10) << time
					  << "\tEdges: " << std::setw(10) << edgeTraversed
					  << "\tsource: " << Sources[0] << std::endl;

		if (CHECK_RESULT)
		{
			dist_t *Dist = dynamic_cast<GraphSSSP &>(graph).BellmanFord_Result();
			if (CUDA_DEBUG >= 3)
			{
				printExt::host::printArray(Dist, graph.V, "host_distance:");
				int *devArray = new int[graph.V];
				hipMemcpy(devArray, devDistances, graph.V * sizeof(int), hipMemcpyDeviceToHost);
				printExt::host::printArray(devArray, graph.V, "cuda_distance:");
				delete[] devArray;
			}

			cuda_util::Compare(Dist, devDistances, graph.V, distanceCompare);
			printf("the %d test is ok", i);
					dynamic_cast<GraphSSSP &>(graph).BellmanFord_Queue_reset();
			//TODO chl
		}

		/*std::cout << "reset start" << std::endl;

	   std::cout << "reset end" << std::endl;*/
	}
	if (CHECK_RESULT)
		{
				dynamic_cast<GraphSSSP &>(graph).BellmanFord_Queue_end();
		}

	std::cout << std::endl
			  << "\tNumber of TESTS: " << N_OF_TESTS << std::endl
			  << "\t      Avg. Time: " << totalTime / N_OF_TESTS << " ms" << std::endl
			  << "\t     Avg. MTEPS: " << totalEdges / (totalTime * 1000) << std::endl
			  << "\t    maxFrontier: " << maxFrontier << std::endl
			  << std::endl;
}

inline void HBFGraph::DynamicVirtualWarpForLast(const int F1Size, const int level)
{
	int size = numeric::log2(RESIDENT_THREADS / F1Size);
	if (MIN_VW >= 1 && size < LOG2<MIN_VW>::value)
		size = LOG2<MIN_VW>::value;
	if (MAX_VW >= 1 && size > LOG2<MAX_VW>::value)
		size = LOG2<MAX_VW>::value;

#define funB(a) kernels::chl_kernel<(a), false>         \
	<<<_Div(graph.V, (BLOCKDIM / (a)) * ITEM_PER_WARP), \
	   BLOCKDIM,                                        \
	   SMem_Per_Block<char, BLOCKDIM>::value>>>(devOutNodes, devOutEdges, devDistances, devF1, devF2, F1Size, level);

	def_SWITCHB(size);
#undef funB
}

inline void HBFGraph::DynamicVirtualWarp(const int F1Size, const int level)
{
	int size = numeric::log2(RESIDENT_THREADS / F1Size);
	if (MIN_VW >= 1 && size < LOG2<MIN_VW>::value)
		size = LOG2<MIN_VW>::value;
	if (MAX_VW >= 1 && size > LOG2<MAX_VW>::value)
		size = LOG2<MAX_VW>::value;
//printf("VW_SIZE:%d\n", size);

/*//#define fun(a)	BF_Kernel1<(a), false>\
    //                    <<<std::min(_DIV(graph.V, BLOCKDIM), 96), BLOCKDIM, SM_DYN>>>\
    //					(devOutNode, devOutEdge, devDistance, devF1, devF2,  F1Size, level);*/

//TODO --chl
#define fun(a) kernels::BF_Kernel1<(a), false>          \
	<<<_Div(graph.V, (BLOCKDIM / (a)) * ITEM_PER_WARP), \
	   BLOCKDIM,                                        \
	   SMem_Per_Block<char, BLOCKDIM>::value>>>(devOutNodes, devOutEdges, devDistances, devF1, devF2, F1Size, level);

	def_SWITCH(size);

#undef fun
}
